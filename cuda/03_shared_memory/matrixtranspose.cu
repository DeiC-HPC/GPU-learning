#include "hip/hip_runtime.h"
#include "matrixprint.h"
#include <stdlib.h>
#include <time.h>
#define T 16

using namespace std;

__global__ void matrixtranspose(const int *a, int *trA, ushort colsA,
                                ushort rowsA) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < colsA && i < rowsA) {
    trA[j * rowsA + i] = a[i * colsA + j];
  }
}

int main() {
  int height = 23000;
  int width = 23000;
  int dimx = ceil(((float)height) / T);
  int dimy = ceil(((float)width) / T);
  dim3 block(T, T, 1), grid(dimx, dimy, 1);
  int memsize = width * height * sizeof(int);
  clock_t start, end;

  int *a, *trA;
  hipMallocManaged(&a, memsize);
  hipMallocManaged(&trA, memsize);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      a[i * width + j] = i * width + j;
    }
  }

  matrixtranspose<<<grid, block>>>(a, trA, width, height);
  hipDeviceSynchronize();

  start = clock();

  matrixtranspose<<<grid, block>>>(a, trA, width, height);
  hipDeviceSynchronize();

  end = clock();

  printf("Elapsed time: %f\n", (double)(end - start) / CLOCKS_PER_SEC);

  printMatrix(trA, height, width);

  return 0;
}
