#include "hip/hip_runtime.h"
#include<stdlib.h>
#include "matrixprint.h"
#define T 16

using namespace std;


__global__ void matrixtranspose(
    const int *A,
    int *trA,
    int colsA,
    int rowsA)
{
    __shared__ int tile[T][T+1];

    int x = blockIdx.x * T + threadIdx.x;
    int y = blockIdx.y * T + threadIdx.y;
    if (y < rowsA && x < colsA) {
        tile[threadIdx.y][threadIdx.x] = A[y*colsA+x];
    }

    __syncthreads();

    x = blockIdx.y * T + threadIdx.x;
    y = blockIdx.x * T + threadIdx.y;
    if (y < rowsA && x < colsA) {
        trA[y*rowsA + x] = tile[threadIdx.x][threadIdx.y];
    }
}

int main() {
    int height = 23000;
    int width = 23000;
    int dimx = ceil(((float)height)/T);
    int dimy = ceil(((float)width)/T);
    dim3 block(T, T, 1), grid(dimx, dimy, 1);
    int memsize = width*height*sizeof(int);
    clock_t start,end;

    int *a, *trA;
    hipMallocManaged(&a, memsize);
    hipMallocManaged(&trA, memsize);

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            a[i*width+j] = i*width+j;
        }
    }

    matrixtranspose<<<grid, block>>>(
            a,
            trA,
            width,
            height);
    hipDeviceSynchronize();

    start = clock();

    matrixtranspose<<<grid, block>>>(
            a,
            trA,
            width,
            height);
    hipDeviceSynchronize();

    end = clock();

    printf("Elapsed time: %f\n", (double)(end-start)/CLOCKS_PER_SEC);

    printMatrix(trA, height, width);

    return 0;
}
