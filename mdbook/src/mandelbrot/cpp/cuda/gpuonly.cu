#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"
#include <fstream>
#include <iostream>
#include <timer.h>

#define T 32

using namespace std;

/* ANCHOR: mandelbrot */
__global__ void mandelbrot(int *res, ushort width, ushort height, float xmin,
                           float xmax, float ymin, float ymax,
                           ushort max_iterations) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  float widthf = width - 1.0f;
  float heightf = height - 1.0f;

  if (x < width && y < height) {
    hipFloatComplex z =
        make_hipFloatComplex(xmin + ((xmax - xmin) * x / widthf),
                            ymax - ((ymax - ymin) * y / heightf));
    hipFloatComplex c = z;

    for (int i = 0; i < max_iterations; i++) {
      if (z.x * z.x + z.y * z.y <= 4.0f) {
        res[y * width + x] = i + 1;
        z = hipCmulf(z, z);
        z = hipCaddf(z, c);
      }
    }
  }
}
/* ANCHOR_END: mandelbrot */

int main() {
  int width = 1000;
  int height = 1000;
  int maxiterations = 100;
  float ymin = -2.0;
  float ymax = 2.0;
  float xmin = -2.5;
  float xmax = 1.5;
  int dimx = ceil(((float)width) / T);
  int dimy = ceil(((float)height) / T);
  dim3 block(T, T, 1), grid(dimx, dimy, 1);
  int resmemsize = width * height * sizeof(int);

  int *res = new int[width * height];
  int *res_device;
  hipMalloc((void **)&res_device, resmemsize);

  timer time;
  mandelbrot<<<grid, block>>>(res_device, width, height, xmin, xmax, ymin, ymax,
                              maxiterations);
  hipDeviceSynchronize();

  hipMemcpy(res, res_device, resmemsize, hipMemcpyDeviceToHost);
  cout << "Elapsed time: " << time.getTime() << endl;

  ofstream file;
  file.open("mandelbrot_gpuonly.csv");

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      if (j != 0) {
        file << ",";
      }
      file << res[i * width + j];
    }
    file << endl;
  }

  file.close();

  return 0;
}
